
#include <hip/hip_runtime.h>
/* this file is for the GPU version thrust call */

#ifdef PARGEMSLR_CUDA

#include "../utils/memory.hpp"
#include "../utils/parallel.hpp"
#include "../utils/utils.hpp"
#include "../vectors/int_vector.hpp"
#include "../vectors/sequential_vector.hpp"
#include "matrix.hpp"
#include "csr_matrix.hpp"
#include "matrixops.hpp"

#include <cstring>
#ifdef PARGEMSLR_CUDA
#include <cuda_runtime.h>
#include "cublas_v2.h"
#include "cusparse.h"
#endif

namespace pargemslr
{  
   template <typename T>
   int CsrMatrixClass<T>::Eye()
   {
      /* don't need to worry for the host version */
      int   n, err = 0;
      int   location = this->GetDataLocation();
      
      PARGEMSLR_CHKERR( this->_ncols != this->_nrows);
      
      /* change to hold value */
      this->_isholdingdata = true;
      
      if(this->_ncols == 0)
      {
         return PARGEMSLR_SUCCESS;
      }
      
      n = this->_ncols;
      
      this->_nnz = n;
      
      /* create I, J, and A */
      err = this->_i_vec.UnitPerm(); PARGEMSLR_CHKERR(err);
      err = this->_j_vec.Setup(n, location, false); PARGEMSLR_CHKERR(err);
      err = this->_j_vec.UnitPerm(); PARGEMSLR_CHKERR(err);
      err = this->_a_vec.Setup(n, location, false); PARGEMSLR_CHKERR(err);
      err = this->_a_vec.Fill(1.0); PARGEMSLR_CHKERR(err);
      
      if( location == kMemoryDevice || location == kMemoryUnified)
      {
         CsrMatrixCreateCusparseSpMat(*this);
      }
      
      return err;
      
   }
   template int CsrMatrixClass<float>::Eye();
   template int CsrMatrixClass<double>::Eye();
   template int CsrMatrixClass<complexs>::Eye();
   template int CsrMatrixClass<complexd>::Eye();
   
   template <typename T>
   int CsrMatrixClass<T>::MoveData( const int &location)
   {
      int loc_from = this->GetDataLocation();
      
      if(location == loc_from)
      {
         return PARGEMSLR_SUCCESS;
      }
      
      if(location == kMemoryDevice || location == kMemoryUnified)
      {
         this->SortRow();
      }
      
      this->_i_vec.MoveData(location);
      this->_j_vec.MoveData(location);
      if(this->_isholdingdata)
      {
         this->_a_vec.MoveData(location);
      }
      
      if( location == kMemoryDevice || location == kMemoryUnified)
      {
         CsrMatrixCreateCusparseSpMat(*this);
      }
      
      return PARGEMSLR_SUCCESS;
   }
   template int CsrMatrixClass<float>::MoveData( const int &location);
   template int CsrMatrixClass<double>::MoveData( const int &location);
   template int CsrMatrixClass<complexs>::MoveData( const int &location);
   template int CsrMatrixClass<complexd>::MoveData( const int &location);
}
#endif
